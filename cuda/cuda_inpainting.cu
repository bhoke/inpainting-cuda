#include "hip/hip_runtime.h"

#include "cuda_inpainting.h"
#include <hip/hip_runtime.h>
#include <vector>
#include <hip/hip_runtime_api.h>

using namespace std;
using namespace cv;

const float CudaInpainting::RANGE_RATIO = 2.0f;
const int CudaInpainting::NODE_SIZE = 8;
const int CudaInpainting::PATCH_SIZE = NODE_SIZE * 2;

const float CudaInpainting::FULL_MSG = CudaInpainting::PATCH_SIZE * CudaInpainting::PATCH_SIZE * 255 * 255 * 3 / 2.0f;

// take one arguement as the input image file
CudaInpainting::CudaInpainting(const char *path) {
	initFlag = false;
	image = imread(path, IMREAD_COLOR);
	imageData = nullptr;
	if(!image.data) {
		cout << "Image loading failed" << endl;
		return;
	}
	image.convertTo(image, CV_32FC3);
	
	// copy the image data to float array
	imageData = new float[3 * image.cols * image.rows];
	hipMalloc((void**)&deviceImageData, sizeof(float) * 3 * image.cols * image.rows);
	if(!imageData) {
		cout << "Memory allocation failed" << endl;
		hipFree(deviceImageData);
		return;
	}
	for(int y = 0; y < image.rows; ++y) {
		for(int x = 0; x < image.cols; ++x) {
			Vec3f vec = image.at<Vec3f>(y, x);
			imageData[3 * image.cols * y + 3 * x] = vec[0];
			imageData[3 * image.cols * y + 3 * x + 1] = vec[1];
			imageData[3 * image.cols * y + 3 * x + 2] = vec[2];
		}
	}

	// copy the raw data to the GPU
	hipMemcpy(deviceImageData, imageData, sizeof(float) * 3 * image.cols * image.rows, hipMemcpyHostToDevice);
	imgWidth = image.cols;
	imgHeight = image.rows;

	// initialize all the ointers
	choiceList = nullptr;
	nodeTable = nullptr;
	patchList = nullptr;
	
	devicePatchList = nullptr;
	deviceSSDTable = nullptr;
	deviceNodeTable = nullptr;
	deviceMsgTable = nullptr;
	deviceFillMsgTable = nullptr;
	deviceEdgeCostTable = nullptr;
	deviceChoiceList = nullptr;
}

// destructor for the CudaInpainting
CudaInpainting::~CudaInpainting() {
	if(imageData) {
		delete imageData;
		hipFree(deviceImageData);
	}
	if(choiceList)
		delete choiceList;
	if(patchList)
		delete patchList;
	if(nodeTable)
		delete nodeTable;

	if(devicePatchList)
		hipFree(devicePatchList);
	if(deviceNodeTable)
		hipFree(deviceNodeTable);
	if(deviceMsgTable)
		hipFree(deviceMsgTable);
	if(deviceFillMsgTable)
		hipFree(deviceFillMsgTable);
	if(deviceEdgeCostTable)
		hipFree(deviceEdgeCostTable);
	if(deviceChoiceList)
		hipFree(deviceChoiceList);
}

// GPU function to copy memory
__global__ void deviceCopyMem(float *src, float *dst, int elem) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x,
	    totalSize = blockDim.x * gridDim.x;
	for(int i = idx; i < elem; i += totalSize) {
		dst[i] = src[i];
		//printf("%d .  %f <= %f\n", i, dst[i], src[i]);
	}
}

// the main function for the image inpainting procedure
bool CudaInpainting::Inpainting(int x,int y, int width, int height, int iterTime) {
	Patch patch(x, y, width, height);
	// first generate the rounded up patch
	patch.round();
	maskPatch = patch;

	// generate the candidate patches list
	GenPatches();
	hipDeviceSynchronize();
	if(patchListSize == 0)
		return true;

	// begin to calculate toe SSD table
	CalculateSSDTable();
	hipDeviceSynchronize();

	// before the iterations, we need to initialize the node table and the message
	InitNodeTable();
	deviceCopyMem<<<dim3(32,1), dim3(1024,1)>>>(deviceFillMsgTable, deviceMsgTable, nodeWidth * nodeHeight * DIR_COUNT * patchListSize);
	hipDeviceSynchronize();

	for(int i = 0; i < iterTime; i++) {
		RunIteration(i);
		deviceCopyMem<<<dim3(32,1), dim3(1024,1)>>>(deviceFillMsgTable, deviceMsgTable, nodeWidth * nodeHeight * DIR_COUNT * patchListSize);
		hipDeviceSynchronize();
		cout<<"ITERATION "<<i<<endl;
	}

	// calculate the best patch for each node
	SelectPatch();

	// fill the patch into the original image 
	FillPatch();

	// use a median filter to make the edge between two patches more smooth
	Rect rect(maskPatch.x - NODE_SIZE, maskPatch.y - NODE_SIZE, 
			maskPatch.width + 2 * NODE_SIZE, maskPatch.height + 2 * NODE_SIZE);
	Mat subMat = image(rect);
	Mat matArr[3];
	// split into multiple color channel
	split(subMat, matArr);
	
	for(int i = 0; i < 3; i++) {
		matArr[i].convertTo(matArr[i], CV_8U);
		medianBlur(matArr[i], matArr[i], 3);
		//GaussianBlur(matArr[i], matArr[i], Size(9,9), 0, 0);
		matArr[i].convertTo(matArr[i], CV_32F);
	}
	// merge the color changes into a color image
	merge(matArr, 3, subMat);

	return true;
}

Mat CudaInpainting::GetImage() {
	return image;
}

// // private functions
//  Patch round(Patch p) {
// 	Patch res;
// 	res.x = (p.x / NODE_SIZE) * NODE_SIZE;
// 	res.y = (p.y / NODE_SIZE) * NODE_SIZE;
// 	res.width = (p.x + p.width +NODE_SIZE - 1) / NODE_SIZE * NODE_SIZE - res.x;
// 	res.height = (p.y + p.height + NODE_SIZE - 1) / NODE_SIZE * NODE_SIZE - res.y;
// 	return res;
// }


// to judge if two given patches have overlap region
bool CudaInpainting::hasOverlap(Patch& p1, Patch& p2) {
	int mLX = p1.x < p2.x ? p2.x : p1.x,
	    mRX = (p1.x+p1.width) < (p2.x+p2.width) ? (p1.x+p1.width) : (p2.x+p2.width),
	    mTY = p1.y < p2.y ? p2.y : p1.y,
	    mBY = (p1.y+p1.height) < (p2.y+p2.height) ? (p1.y+p1.height) : (p2.y+p2.height);
	return mRX > mLX && mBY > mTY;
}

// generate the patches list
void CudaInpainting::GenPatches() {
	vector<Patch> tmpPatchList;
	Patch p = maskPatch;
	cout << "x=" << p.x << " y=" << p.y << " width=" << p.width << " height=" << p.height << endl;
	int hh = image.rows / NODE_SIZE,
	    ww = image.cols / NODE_SIZE;
	float midX = p.x + p.width / 2,
	      midY = p.y + p.height / 2;
	for(int i = 1; i <= hh; i++) {
		for(int j = 1; j <= ww; j++) {
			int cX, cY;
			float fcx = j * NODE_SIZE, fcy = i * NODE_SIZE;
			cY = i * NODE_SIZE - NODE_SIZE;
			cX = j * NODE_SIZE - NODE_SIZE;
			if(!(fabsf(fcx - midX) * 2 / p.width < RANGE_RATIO && fabsf(fcy - midY) * 2 / p.height < RANGE_RATIO))
				continue;
			if(image.rows - cY < PATCH_SIZE || image.cols - cX < PATCH_SIZE)
				continue;
			Patch cur(cX, cY, PATCH_SIZE, PATCH_SIZE);
			if(!hasOverlap(cur, p))
				tmpPatchList.push_back(cur);
		}
	}
	patchListSize = tmpPatchList.size();
	if(tmpPatchList.size() == 0)
		return;
	hipMalloc((void**)&devicePatchList, sizeof(Patch) * tmpPatchList.size());
	patchList = new Patch[tmpPatchList.size()];
	if(!patchList) {
		cout << "NULL patchList! exit"<< endl;
		exit(-1);
	}
	for(int i = 0; i < tmpPatchList.size(); i++) {
		patchList[i] = tmpPatchList[i];
	}
	// copy the generated patches list to the GPU global memory
	hipMemcpy(devicePatchList, patchList, sizeof(Patch) * tmpPatchList.size(), hipMemcpyHostToDevice);
	cout << "GenPatch done, " << patchListSize << " patches generated" << endl;
	int idx = 23;
	cout << "Patch => " << idx << " : x=" << patchList[idx].x << " y=" << patchList[idx].y << endl;
	cout << "devicePatchList=" << devicePatchList << endl;
}

// a helper to get the message position in the message table
__device__ inline int getMsgIdx(int x, int y, CudaInpainting::EDIR dir, int l, int ww, int hh, int len) {
	return y * ww * CudaInpainting::EDIR::DIR_COUNT * len + x * CudaInpainting::EDIR::DIR_COUNT * len +
		dir * len + l;
}

// a helper to get the edge cost position in the message table
__device__ inline int getEdgeCostIdx(int x, int y, int l, int ww, int hh, int len) {
	return y * ww * len  + x * len + l;
}

// calculate the SSD table on GPU
__global__ void deviceCalculateSSDTable(float *dImg, int ww, int hh, Patch *pl, CudaInpainting::SSDEntry *dSSDTable) {
	int len = gridDim.x;
	const int patchSize = CudaInpainting::PATCH_SIZE * CudaInpainting::PATCH_SIZE;	
	__shared__ float pixels[CudaInpainting::PATCH_SIZE][CudaInpainting::PATCH_SIZE][3];
	for(int i = threadIdx.x; i < patchSize; i += blockDim.x) {
		int yy = i / CudaInpainting::PATCH_SIZE, xx = i % CudaInpainting::PATCH_SIZE;
		int iyy = pl[blockIdx.x].y + yy, ixx = pl[blockIdx.x].x + xx;
		pixels[yy][xx][0] = dImg[iyy * ww * 3 + ixx * 3];
		pixels[yy][xx][1] = dImg[iyy * ww * 3 + ixx * 3 + 1];
		pixels[yy][xx][2] = dImg[iyy * ww * 3 + ixx * 3 + 2];
	}

	__syncthreads();

	for(int i = threadIdx.x; i < len; i += blockDim.x) {
		int px = pl[i].x, py = pl[i].y;
		for(int j = 0; j < CudaInpainting::EPOS_COUNT; j++) {
			float res = 0;
			int WW, HH;
			int pxx, pyy;
			switch(j) {
				case CudaInpainting::UP_DOWN:
					WW = CudaInpainting::PATCH_SIZE;
					HH = CudaInpainting::NODE_SIZE;
					for(int dy = 0; dy < HH; ++dy) {
						for(int dx = 0; dx < WW; ++dx) {
							pxx = px + dx;
							pyy = py + dy;
							float rr = pixels[dy + CudaInpainting::NODE_SIZE][dx][0] - dImg[pyy * ww * 3 + pxx * 3],
							      gg = pixels[dy + CudaInpainting::NODE_SIZE][dx][1] - dImg[pyy * ww * 3 + pxx * 3 + 1],
							      bb = pixels[dy + CudaInpainting::NODE_SIZE][dx][2] - dImg[pyy * ww * 3 + pxx * 3 + 2];
							rr *= rr;
							gg *= gg;
							bb *= bb;
							res += rr + gg + bb;
						}
					}
					break;
				case CudaInpainting::DOWN_UP:
					WW = CudaInpainting::PATCH_SIZE;
					HH = CudaInpainting::NODE_SIZE;
					for(int dy = 0; dy < HH; ++dy) {
						for(int dx = 0; dx < WW; ++dx) {
							pxx = px + dx;
							pyy = py + dy + CudaInpainting::NODE_SIZE;
							float rr = pixels[dy][dx][0] - dImg[pyy * ww * 3 + pxx * 3],
							      gg = pixels[dy][dx][1] - dImg[pyy * ww * 3 + pxx * 3 + 1],
							      bb = pixels[dy][dx][2] - dImg[pyy * ww * 3 + pxx * 3 + 2];
							rr *= rr;
							gg *= gg;
							bb *= bb;
							res += rr + gg + bb;
						}
					}
					break;
				case CudaInpainting::RIGHT_LEFT:
					WW = CudaInpainting::NODE_SIZE;
					HH = CudaInpainting::PATCH_SIZE;
					for(int dy = 0; dy < HH; ++dy) {
						for(int dx = 0; dx < WW; ++dx) {
							pxx = px + dx + CudaInpainting::NODE_SIZE;
							pyy = py + dy;
							float rr = pixels[dy][dx][0] - dImg[pyy * ww * 3 + pxx * 3],
							      gg = pixels[dy][dx][1] - dImg[pyy * ww * 3 + pxx * 3 + 1],
							      bb = pixels[dy][dx][2] - dImg[pyy * ww * 3 + pxx * 3 + 2];
							rr *= rr;
							gg *= gg;
							bb *= bb;
							res += rr + gg + bb;
						}
					}
					break;
				case CudaInpainting::LEFT_RIGHT:
					WW = CudaInpainting::NODE_SIZE;
					HH = CudaInpainting::PATCH_SIZE;
					for(int dy = 0; dy < HH; ++dy) {
						for(int dx = 0; dx < WW; ++dx) {
							pxx = px + dx;
							pyy = py + dy;
							float rr = pixels[dy][dx + CudaInpainting::NODE_SIZE][0] - dImg[pyy * ww * 3 + pxx * 3],
							      gg = pixels[dy][dx + CudaInpainting::NODE_SIZE][1] - dImg[pyy * ww * 3 + pxx * 3 + 1],
							      bb = pixels[dy][dx + CudaInpainting::NODE_SIZE][2] - dImg[pyy * ww * 3 + pxx * 3 + 2];
							rr *= rr;
							gg *= gg;
							bb *= bb;
							res += rr + gg + bb;
						}
					}
					break;
			}
			dSSDTable[blockIdx.x * len + i].data[j] = res;
		}
	}
}

void CudaInpainting::CalculateSSDTable() {
	hipMalloc((void**)&deviceSSDTable, sizeof(SSDEntry) * patchListSize * patchListSize);
	if(devicePatchList && deviceSSDTable) {
		cout << "Calculate SSDTable" << endl;
		int len = PATCH_SIZE * PATCH_SIZE;
		if(len > 1024)
			len = 1024;
		cout << "CUDA PARAM: " << patchListSize << "=>" << len << endl;
		deviceCalculateSSDTable<<<dim3(patchListSize, 1), dim3(len, 1)>>>(deviceImageData, imgWidth, imgHeight, devicePatchList, deviceSSDTable);
	}
}

__device__ float deviceCalculateSSD(float *dImg, int w, int h, Patch p1, Patch p2, CudaInpainting::EPOS pos) {
	float res = 0;
	int ww, hh;
	int p1x, p1y, p2x, p2y;
	switch(pos) {
		case CudaInpainting::UP_DOWN:
		case CudaInpainting::DOWN_UP:
			if(pos == CudaInpainting::UP_DOWN) {
				p1x = p1.x;
				p1y = p1.y;
				p2x = p2.x;
				p2y = p2.y;
			} else {
				p1x = p2.x;
				p1y = p2.y;
				p2x = p1.x;
				p2y = p1.y;
			}
			ww = CudaInpainting::PATCH_SIZE;
			hh = CudaInpainting::NODE_SIZE;
			for(int i = 0; i < hh; ++i) {
				for(int j = 0; j < ww; ++j) {
					float rr = dImg[(p1y + CudaInpainting::NODE_SIZE + i) * w * 3 + (p1x + j) * 3] - dImg[(p2y + i) * w * 3 + (p2x + j) * 3],
					      gg = dImg[(p1y + CudaInpainting::NODE_SIZE + i) * w * 3 + (p1x + j
) * 3 + 1] - dImg[(p2y + i) * w * 3 + (p2x + j) * 3 + 1],
					      bb = dImg[(p1y + CudaInpainting::NODE_SIZE + i) * w * 3 + (p1x + j
) * 3 + 2] - dImg[(p2y + i) * w * 3 + (p2x + j) * 3 + 2];
					rr *= rr;
					gg *= gg;
					bb *= bb;
					res += rr + gg + bb;
				}
			}
			break;
		case CudaInpainting::LEFT_RIGHT:
		case CudaInpainting::RIGHT_LEFT:
			if(pos == CudaInpainting::LEFT_RIGHT) {
				p1x = p1.x;
				p1y = p1.y;
				p2x = p2.x;
				p2y = p2.y;
			} else {
				p1x = p2.x;
				p1y = p2.y;
				p2x = p1.x;
				p2y = p1.y;
			}
			ww = CudaInpainting::NODE_SIZE;
			hh = CudaInpainting::PATCH_SIZE;
			for(int i = 0; i < hh; ++i) {
				for(int j = 0; j < ww; ++j) {
					float rr = dImg[(p1y + i) * w * 3 + (p1x + CudaInpainting::NODE_SIZE + j) * 3] - dImg[(p2y + i) * w * 3 + (p2x + j) * 3],
					      gg = dImg[(p1y + i) * w * 3 + (p1x + CudaInpainting::NODE_SIZE + j) * 3 + 1] - dImg[(p2y + i) * w * 3 + (p2x + j) * 3 + 1],
					      bb = dImg[(p1y + i) * w * 3 + (p1x + CudaInpainting::NODE_SIZE + j) * 3 + 2] - dImg[(p2y + i) * w * 3 + (p2x + j) * 3 + 2];
					rr *= rr;
					gg *= gg;
					bb *= bb;
					res += rr + gg + bb;
				}
			}
			break;
	}
	return res;
}

// initialize the coordinates of node in table
__global__ void deviceInitFirst(CudaInpainting::Node* dNodeTable, Patch p) {
	int ww = gridDim.x;
	dNodeTable[ww * threadIdx.x + blockIdx.x].x = p.x + blockIdx.x * CudaInpainting::NODE_SIZE;
	dNodeTable[ww * threadIdx.x + blockIdx.x].y = p.y + threadIdx.x * CudaInpainting::NODE_SIZE;
}

// the constructor of patch on GPU
__device__ Patch::Patch(int ww, int hh) {
	width = ww;
	height = hh;
}

// the initialize node table on GPU
__global__ void deviceInitNodeTable(float *dImg, int w, int h, Patch p, CudaInpainting::Node* dNodeTable, float *dMsgTable, float *dEdgeCostTable, Patch *dPatchList, int len) {
	int hh = gridDim.y, ww = gridDim.x;

	for(int i = threadIdx.x; i < len; i += blockDim.x * blockDim.y) {
		// initialize the message with the very large values
		dMsgTable[getMsgIdx(blockIdx.x, blockIdx.y, CudaInpainting::DIR_UP, i, ww, hh, len)] = CudaInpainting::FULL_MSG;
		dMsgTable[getMsgIdx(blockIdx.x, blockIdx.y, CudaInpainting::DIR_DOWN, i, ww, hh, len)] = CudaInpainting::FULL_MSG;
		dMsgTable[getMsgIdx(blockIdx.x, blockIdx.y, CudaInpainting::DIR_LEFT, i, ww, hh, len)] = CudaInpainting::FULL_MSG;
		dMsgTable[getMsgIdx(blockIdx.x, blockIdx.y, CudaInpainting::DIR_RIGHT, i, ww, hh, len)] = CudaInpainting::FULL_MSG;

		// initialize the edge cost 
		float val = 0;
		Patch curPatch(CudaInpainting::PATCH_SIZE, CudaInpainting::PATCH_SIZE);
		
		// to judge if the current node is on the edge of the node table
		if(((blockIdx.y == 0 || blockIdx.y == hh - 1) && (/*blockIdx.x >= 0 && */blockIdx.x <= ww - 1 )) ||
					((blockIdx.x == 0 || blockIdx.x == ww - 1) && (/*blockIdx.y >= 0 && */blockIdx.y <= hh - 1))) {
			int nodeIdx = ww * blockIdx.y + blockIdx.x;
			int valCount = 0;
			if(blockIdx.x == 0) {
				curPatch.x = dNodeTable[nodeIdx].x - CudaInpainting::PATCH_SIZE;
				curPatch.y = dNodeTable[nodeIdx].y - CudaInpainting::NODE_SIZE;
				val += deviceCalculateSSD(dImg, w, h, curPatch, dPatchList[i], CudaInpainting::LEFT_RIGHT);
				++valCount;
			} else {
				curPatch.x = dNodeTable[nodeIdx].x;
				curPatch.y = dNodeTable[nodeIdx].y - CudaInpainting::NODE_SIZE;
				val += deviceCalculateSSD(dImg, w, h, dPatchList[i], curPatch, CudaInpainting::LEFT_RIGHT);
				++valCount;
			}
			if(blockIdx.y == 0) {
				curPatch.x = dNodeTable[nodeIdx].x - CudaInpainting::NODE_SIZE;
				curPatch.y = dNodeTable[nodeIdx].y - CudaInpainting::PATCH_SIZE;
				val += deviceCalculateSSD(dImg, w, h, curPatch, dPatchList[i], CudaInpainting::UP_DOWN);
				++valCount;
			} else {
				curPatch.x = dNodeTable[nodeIdx].x - CudaInpainting::NODE_SIZE;
				curPatch.y = dNodeTable[nodeIdx].y;
				val += deviceCalculateSSD(dImg, w, h, dPatchList[i], curPatch, CudaInpainting::UP_DOWN);
				++valCount;
			}
			val /= valCount;
		}
		if(val < 0.5f)
			val = CudaInpainting::FULL_MSG;
		dEdgeCostTable[getEdgeCostIdx(blockIdx.x, blockIdx.y, i, ww, hh, len)] = val;
	}
}


// wrap for the initialization of the node table
void CudaInpainting::InitNodeTable() {
	nodeHeight = maskPatch.height / NODE_SIZE + 1;
	nodeWidth = maskPatch.width / NODE_SIZE + 1;
	cout << "NodeTable => width=" << nodeWidth << " height=" << nodeHeight << endl;
	int totalElement = nodeWidth * nodeHeight * DIR_COUNT * patchListSize;
	cout << hipGetErrorString(hipMalloc((void**)&deviceNodeTable, sizeof(Node) * nodeWidth * nodeHeight)) << endl;
	cout << hipGetErrorString(hipMalloc((void**)&deviceMsgTable, sizeof(float) * totalElement)) << endl;
	cout << hipGetErrorString(hipMalloc((void**)&deviceFillMsgTable, sizeof(float) * totalElement)) << endl;
	cout << hipGetErrorString(hipMalloc((void**)&deviceEdgeCostTable, sizeof(float) * nodeWidth * nodeHeight * patchListSize)) << endl;
	if(deviceNodeTable && deviceMsgTable && deviceFillMsgTable && deviceEdgeCostTable) {
		cout << "Initialize the Node Table and Message Table" << endl;
		// initialize node table
		deviceInitFirst<<<dim3(nodeWidth, 1), dim3(nodeHeight,1)>>>(deviceNodeTable, maskPatch);

		// initialize the messages in the node table
		deviceInitNodeTable<<<dim3(nodeWidth, nodeHeight), dim3(512,1)>>>(deviceImageData, imgWidth, imgHeight, maskPatch, deviceNodeTable, deviceFillMsgTable, deviceEdgeCostTable, devicePatchList, patchListSize);
	} else {
		cout << " Failed to hipMalloc" << endl;
	}

	// initialize the node table on CPU
	nodeTable = new Node[nodeWidth * nodeHeight];
	if(nodeTable) {
		for(int i = 0; i < nodeHeight; ++i) {
			for(int j = 0; j < nodeWidth; ++j) {
				nodeTable[i * nodeWidth + j].x = maskPatch.x + j * NODE_SIZE;
				nodeTable[i * nodeWidth + j].y = maskPatch.y + i * NODE_SIZE;
			}
		}
	}
}

// the iteration function which will be run on GPU
__global__ void deviceIteration(CudaInpainting::SSDEntry *dSSDTable, float *dEdgeCostTable, Patch *dPatchList, int len, float *dMsgTable, float *dFillMsgTable, int times) {
	int hh = gridDim.y, ww = gridDim.x, i = blockIdx.y, j = blockIdx.x;
	float aroundMsg, msgCount, matchFactor;
	float msgFactor = 0.8f;
	matchFactor = 1.2f;
	msgCount = msgFactor * 3 + matchFactor + 1;
	// each thread handle one patch in all directions
	for(int ll = threadIdx.x; ll < len; ll += blockDim.x) {
		// use register to optimize the running time
		float up_val, down_val, left_val, right_val;
		// up
		if(i != 0) {
			int targetIdx = getMsgIdx(j, i, CudaInpainting::DIR_UP, ll, ww, hh, len);
			up_val = dFillMsgTable[targetIdx];
		}
		// down
		if(i != hh - 1) {
			int targetIdx = getMsgIdx(j, i, CudaInpainting::DIR_DOWN, ll, ww, hh, len);
			down_val = dFillMsgTable[targetIdx];
		}
		// left
		if(j != 0) {
			int targetIdx = getMsgIdx(j, i, CudaInpainting::DIR_LEFT, ll, ww, hh, len);
			left_val = dFillMsgTable[targetIdx];
		}
		// right
		if(j != ww - 1) {
			int targetIdx = getMsgIdx(j, i, CudaInpainting::DIR_RIGHT, ll, ww, hh, len);
			right_val = dFillMsgTable[targetIdx];
		}

		for(int k = 0; k < len; ++k) {
			float distDiff = 1;
			aroundMsg = 0;
			if(i != 0)
				aroundMsg += dMsgTable[getMsgIdx(j, i - 1, CudaInpainting::DIR_DOWN, k, ww, hh, len)];
			else if(i != hh - 1)
				aroundMsg += dMsgTable[getMsgIdx(j, i + 1, CudaInpainting::DIR_UP, k, ww, hh, len)];
			else if(j != 0)
				aroundMsg += dMsgTable[getMsgIdx(j - 1, i, CudaInpainting::DIR_RIGHT, k, ww, hh, len)];
			else if(j != ww - 1)
				aroundMsg += dMsgTable[getMsgIdx(j + 1, i, CudaInpainting::DIR_LEFT, k, ww, hh, len)];
			else
				aroundMsg += CudaInpainting::FULL_MSG;
			aroundMsg *= msgFactor;
			float edgeVal = dEdgeCostTable[getEdgeCostIdx(j, i, k, ww, hh, len)];
			aroundMsg += edgeVal;
			float val, oldVal;
			// up
			if(i != 0) {
				val = aroundMsg + dSSDTable[k * len + ll].data[CudaInpainting::DOWN_UP] * matchFactor * distDiff;
				val -= dMsgTable[getMsgIdx(j, i - 1, CudaInpainting::DIR_DOWN, k, ww, hh, len)] * msgFactor;
				int targetIdx = getMsgIdx(j, i, CudaInpainting::DIR_UP, ll, ww, hh, len);
				val /= msgCount;
				oldVal = up_val;
				//printf("(%d,%d,-%d) => val=%f\n", j, i, ll, val);
				if(val < oldVal) {
					up_val = val;
				}
			}
			// down
			if(i != hh - 1) {
				val = aroundMsg + dSSDTable[k * len + ll].data[CudaInpainting::UP_DOWN] * matchFactor * distDiff;
				val -= dMsgTable[getMsgIdx(j, i + 1, CudaInpainting::DIR_UP, k, ww, hh, len)] * msgFactor;
				val /= msgCount;
				int targetIdx = getMsgIdx(j, i, CudaInpainting::DIR_DOWN, ll, ww, hh, len);
				oldVal = down_val;
				if(val < oldVal) {
					down_val = val;
				}
			}
			// left
			if(j != 0) {
				val = aroundMsg + dSSDTable[k * len + ll].data[CudaInpainting::RIGHT_LEFT] * matchFactor * distDiff;
				val -= dMsgTable[getMsgIdx(j - 1, i, CudaInpainting::DIR_RIGHT, k, ww, hh, len)] * msgFactor;
				val /= msgCount;
				int targetIdx = getMsgIdx(j, i, CudaInpainting::DIR_LEFT, ll, ww, hh, len);
				oldVal = left_val;
				if(val < oldVal) {
					left_val = val;
				}
			}
			// right
			if(j != ww - 1) {
				val = aroundMsg + dSSDTable[k * len + ll].data[CudaInpainting::LEFT_RIGHT] * matchFactor;
				val -= dMsgTable[getMsgIdx(j + 1, i, CudaInpainting::DIR_LEFT, k, ww, hh, len)] * msgFactor;
				//printf("(%d,%d,-%d) => val=%f oldVal=%f\n", j, i, ll, val, oldVal);
				val /= msgCount;
				int targetIdx = getMsgIdx(j, i, CudaInpainting::DIR_RIGHT, ll, ww, hh, len);
				oldVal = right_val;
				if(val < oldVal) {
					right_val = val;
				}
			}
		}
		// up
		if(i != 0) {
			int targetIdx = getMsgIdx(j, i, CudaInpainting::DIR_UP, ll, ww, hh, len);
			dFillMsgTable[targetIdx] = up_val;
		}
		// down
		if(i != hh - 1) {
			int targetIdx = getMsgIdx(j, i, CudaInpainting::DIR_DOWN, ll, ww, hh, len);
			dFillMsgTable[targetIdx] = down_val;
		}
		// left
		if(j != 0) {
			int targetIdx = getMsgIdx(j, i, CudaInpainting::DIR_LEFT, ll, ww, hh, len);
			dFillMsgTable[targetIdx] = left_val;
		}
		// right
		if(j != ww - 1) {
			int targetIdx = getMsgIdx(j, i, CudaInpainting::DIR_RIGHT, ll, ww, hh, len);
			dFillMsgTable[targetIdx] = right_val;
		}
	}
}

// the wrap function for iteration in Belief Propagation
void CudaInpainting::RunIteration(int times) {
	if(deviceMsgTable && deviceFillMsgTable && deviceSSDTable && deviceEdgeCostTable) {
		int lim = 1024;
		if(patchListSize < lim) {
			lim = patchListSize;
		}
		deviceIteration<<<dim3(nodeWidth, nodeHeight),dim3(lim, 1)>>>(deviceSSDTable, deviceEdgeCostTable,devicePatchList, patchListSize, deviceMsgTable, deviceFillMsgTable, times);
	}
}


// select the best patch for each node on GPU
__global__ void deviceSelectPatch(float *dMsgTable, float *dEdgeCostTable, int *dChoiceList, 
		int ww, int hh,int len) {	
	int xx = blockDim.x * blockIdx.x + threadIdx.x, yy = blockDim.y * blockIdx. y + threadIdx.y;
	if(xx < ww && yy < hh) {
		float maxB = 0;
		int maxIdx = -1;
		for(int k = 0; k < len; ++k) {
			float bl = -dEdgeCostTable[getEdgeCostIdx(xx, yy, k, ww, hh, len)];
			float val;
			if(yy - 1 >= 0) {
				val = dMsgTable[getMsgIdx(xx, yy - 1, CudaInpainting::DIR_DOWN, k, ww, hh, len)];
				if(val > 0)
					bl -= val;
			}
			if(yy + 1 < hh) {
				val = dMsgTable[getMsgIdx(xx, yy + 1, CudaInpainting::DIR_UP, k, ww, hh, len)];
				if(val > 0)
					bl -= val;
			}
			if(xx - 1 >= 0) {
				val = dMsgTable[getMsgIdx(xx - 1, yy, CudaInpainting::DIR_RIGHT, k, ww, hh, len)];
				if(val > 0)
					bl -= val;
			}
			if(xx + 1 < ww) {
				val = dMsgTable[getMsgIdx(xx + 1, yy, CudaInpainting::DIR_LEFT, k, ww, hh, len)];
				if(val > 0)
					bl -= val;
			}
			if(bl > maxB || maxIdx < 0) {
				maxB = bl;
				maxIdx = k;
			}
		}
		//printf("(%d,%d) (%d,%d) => max %f %d\n", ww, hh, xx, yy, maxB, maxIdx);
		dChoiceList[yy * ww + xx] = maxIdx;
	}
}

// the wraper for selecting best patch on GPU
void CudaInpainting::SelectPatch() {
	choiceList = new int[nodeWidth * nodeHeight];
	hipMalloc((void**)&deviceChoiceList, sizeof(float) * nodeWidth * nodeHeight);
	if(choiceList && deviceChoiceList && deviceEdgeCostTable && deviceMsgTable) {
		cout << "Select the Best Patch" << endl;
		deviceSelectPatch<<<dim3((nodeWidth+15)/16, (nodeHeight+15)/16), dim3(16,16)>>>(deviceMsgTable, deviceEdgeCostTable, deviceChoiceList, nodeWidth, nodeHeight, patchListSize);
		hipMemcpy(choiceList, deviceChoiceList, sizeof(int) * nodeWidth * nodeHeight, hipMemcpyDeviceToHost);
	}
}

// the helper to paste the best patch to the specified node
void CudaInpainting::PastePatch(Mat& img, Node& n, Patch& p) {
	int xx = n.x - NODE_SIZE / 2,
	    yy = n.y - NODE_SIZE / 2;
	for(int i = 0; i < p.height / 2; ++i) {
		for(int j = 0; j < p.width / 2; ++j) {
			img.at<Vec3f>(yy + i, xx + j) = img.at<Vec3f>(p.y + NODE_SIZE/2 + i, p.x + NODE_SIZE/2 + j);
		}
	}
}

// paste best patch for all node
void CudaInpainting::FillPatch() {
	int hh = nodeHeight,
	    ww = nodeWidth;
	// just print the result
	for(int i = 0; i < hh; ++i) {
		for(int j = 0; j < ww; ++j) {
			cout<<choiceList[j + i * ww]<<" ";
		}
		cout<<endl;
	}
	for(int i = 0; i < hh; ++i) {
		for(int j = 0; j < ww; ++j) {
			int label = choiceList[j + i * ww];
			if(label >= 0) {
				PastePatch(image, nodeTable[j + i * ww], patchList[label]);
			}
		}
	}
}